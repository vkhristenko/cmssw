#include "RecoLocalCalo/EcalRecAlgos/interface/EcalUncalibRecHitMultiFitAlgo_gpu_new.h"

#include "DataFormats/EcalDigi/interface/EcalDigiCollections.h"
#include "CondFormats/EcalObjects/interface/EcalPedestals.h"
#include "CondFormats/EcalObjects/interface/EcalMGPAGainRatio.h"
#include "CondFormats/EcalObjects/interface/EcalXtalGroupId.h"
#include "CondFormats/EcalObjects/interface/EcalPulseShapes.h"
#include "CondFormats/EcalObjects/interface/EcalPulseCovariances.h"
#include "CondFormats/EcalObjects/interface/EcalSampleMask.h"
#include "CondFormats/EcalObjects/interface/EcalSamplesCorrelation.h"

#include <iostream>
#include <limits>

#include "DataFormats/EcalDigi/interface/EcalDataFrame.h"
#include "RecoLocalCalo/EcalRecAlgos/interface/Common.h"

#include "hip/hip_runtime.h"

#include "AmplitudeComputationCommonKernels.h"
#include "AmplitudeComputationKernelsV1.h"
#include "TimeComputationKernels.h"

//#define DEBUG

//#define ECAL_RECO_CUDA_DEBUG

namespace ecal { namespace multifit {
   
void entryPoint(
        EventInputDataCPU const& eventInputCPU, EventInputDataGPU& eventInputGPU,
        EventOutputDataGPU& eventOutputGPU, EventDataForScratchGPU& scratch,
        ConditionsProducts const& conditions, 
        ConfigurationParameters const& configParameters,
        cuda::stream_t<>& cudaStream) {
    using digis_type = std::vector<uint16_t>;
    using dids_type = std::vector<uint32_t>;
    bool const gainSwitchUseMaxSampleEB = true; // accodring to the cpu setup  //----> hardcoded
    bool const gainSwitchUseMaxSampleEE = false; // accodring to the cpu setup  //----> hardcoded

    uint32_t const offsetForHashes = conditions.offsetForHashes;
    unsigned int totalChannels = eventInputCPU.ebDigis.size() 
        + eventInputCPU.eeDigis.size();
    
    // temporary for recording
    /*hipEvent_t start_event;
    hipEvent_t end_event;
    cudaCheck( hipEventCreate(&start_event) );
    cudaCheck( hipEventCreate(&end_event) );

    cudaCheck (hipEventRecord(start_event, 0) );
    */

    //
    // in what follows we copy eb then ee.
    // offset by size 
    //

    // 
    // copy event data: digis + ids, not really async as vectors have default
    // allocators
    //
    cudaCheck( hipMemcpyAsync(eventInputGPU.digis, 
               eventInputCPU.ebDigis.data().data(),
               eventInputCPU.ebDigis.data().size() * sizeof(digis_type::value_type),
               hipMemcpyHostToDevice,
               cudaStream.id()) );
    cudaCheck( hipMemcpyAsync(eventInputGPU.digis + eventInputCPU.ebDigis.data().size(), 
               eventInputCPU.eeDigis.data().data(),
               eventInputCPU.eeDigis.data().size() * sizeof(digis_type::value_type),
               hipMemcpyHostToDevice,
               cudaStream.id()));

    cudaCheck( hipMemcpyAsync(eventInputGPU.ids, 
               eventInputCPU.ebDigis.ids().data(),
               eventInputCPU.ebDigis.ids().size() * sizeof(dids_type::value_type),
               hipMemcpyHostToDevice,
               cudaStream.id()) );
    cudaCheck (hipMemcpyAsync(eventInputGPU.ids + eventInputCPU.ebDigis.ids().size(), 
               eventInputCPU.eeDigis.ids().data(),
               eventInputCPU.eeDigis.ids().size() * sizeof(dids_type::value_type),
               hipMemcpyHostToDevice,
               cudaStream.id()) );

    // 
    // 1d preparation kernel
    //
    unsigned int nchannels_per_block = 32;
    unsigned int threads_1d = 10 * nchannels_per_block;
    unsigned int blocks_1d = threads_1d > 10*totalChannels 
        ? 1 : (totalChannels*10 + threads_1d - 1) / threads_1d;
    int shared_bytes = nchannels_per_block * EcalDataFrame::MAXSAMPLES * (
        sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(bool) + sizeof(char)
        + sizeof(bool)
    );
    kernel_prep_1d_and_initialize<<<blocks_1d, threads_1d, 
                                    shared_bytes, cudaStream.id()>>>(
        conditions.pulseShapes.values, 
        eventInputGPU.digis, 
        eventInputGPU.ids,
        scratch.samples,
        (SampleVector*)eventOutputGPU.amplitudesAll,
        scratch.gainsNoise,
        conditions.pedestals.mean_x1,
        conditions.pedestals.mean_x12,
        conditions.pedestals.rms_x12,
        conditions.pedestals.mean_x6,
        conditions.gainRatios.gain6Over1,
        conditions.gainRatios.gain12Over6,
        scratch.hasSwitchToGain6,
        scratch.hasSwitchToGain1,
        scratch.isSaturated,
        eventOutputGPU.amplitude,
        eventOutputGPU.chi2,
        eventOutputGPU.pedestal,
        eventOutputGPU.flags,
        scratch.acState,
        scratch.activeBXs,
        offsetForHashes,
        gainSwitchUseMaxSampleEB,
        gainSwitchUseMaxSampleEE,
        totalChannels);
    cudaCheck(hipGetLastError());

    //
    // 2d preparation kernel
    //
    int blocks_2d = totalChannels;
    dim3 threads_2d{10, 10};
    kernel_prep_2d<<<blocks_2d, threads_2d, 0, cudaStream.id()>>>(
        conditions.pulseCovariances.values, 
        scratch.pulse_covariances,
        scratch.gainsNoise,
        eventInputGPU.ids,
        conditions.pedestals.rms_x12,
        conditions.pedestals.rms_x6,
        conditions.pedestals.rms_x1,
        conditions.gainRatios.gain12Over6,
        conditions.gainRatios.gain6Over1,
        conditions.samplesCorrelation.EBG12SamplesCorrelation,
        conditions.samplesCorrelation.EBG6SamplesCorrelation,
        conditions.samplesCorrelation.EBG1SamplesCorrelation,
        conditions.samplesCorrelation.EEG12SamplesCorrelation,
        conditions.samplesCorrelation.EEG6SamplesCorrelation,
        conditions.samplesCorrelation.EEG1SamplesCorrelation,
        scratch.noisecov,
        scratch.pulse_matrix,
        conditions.pulseShapes.values,
        scratch.hasSwitchToGain6,
        scratch.hasSwitchToGain1,
        scratch.isSaturated,
        offsetForHashes);
    cudaCheck(hipGetLastError());
    
    // run minimization kernels
    v1::minimization_procedure(
        eventInputCPU, eventInputGPU, eventOutputGPU,
        scratch, conditions, configParameters, cudaStream);

    if (configParameters.shouldRunTimingComputation) {
        
        //
        // TODO: this guy can run concurrently with other kernels,
        // there is no dependence on the order of execution
        //
        unsigned int threads_time_init = threads_1d;
        unsigned int blocks_time_init = blocks_1d;
        int sharedBytesInit = 2 * threads_time_init * sizeof(SampleVector::Scalar);
        kernel_time_computation_init<<<blocks_time_init, threads_time_init,
                                       sharedBytesInit, cudaStream.id()>>>(
            eventInputGPU.digis, 
            eventInputGPU.ids,
            conditions.pedestals.rms_x12,
            conditions.pedestals.rms_x6,
            conditions.pedestals.rms_x1,
            conditions.pedestals.mean_x12,
            conditions.pedestals.mean_x6,
            conditions.pedestals.mean_x1,
            conditions.gainRatios.gain12Over6,
            conditions.gainRatios.gain6Over1,
            scratch.sample_values,
            scratch.sample_value_errors,
            scratch.ampMaxError,
            scratch.useless_sample_values,
            scratch.pedestal_nums,
            offsetForHashes,
            conditions.sampleMask.getEcalSampleMaskRecordEB(),
            conditions.sampleMask.getEcalSampleMaskRecordEE(),
            totalChannels
        );
        cudaCheck(hipGetLastError());

        // 
        // TODO: small kernel only for EB. It needs to be checked if 
        /// fusing such small kernels is beneficial in here
        //
        // we are running only over EB digis
        // therefore we need to create threads/blocks only for that
        unsigned int const threadsFixMGPA = threads_1d;
        unsigned int const blocksFixMGPA = 
            threadsFixMGPA > 10 * eventInputCPU.ebDigis.size()
                ? 1
                : (10 * eventInputCPU.ebDigis.size() + threadsFixMGPA - 1) 
                    / threadsFixMGPA;
        kernel_time_compute_fixMGPAslew<<<blocksFixMGPA, threadsFixMGPA, 
                                          0, cudaStream.id()>>>(
            eventInputGPU.digis,
            scratch.sample_values,
            scratch.sample_value_errors,
            scratch.useless_sample_values,
            conditions.sampleMask.getEcalSampleMaskRecordEB(),
            totalChannels
        );
        cudaCheck(hipGetLastError());

        //
        // 
        //
        int sharedBytes = EcalDataFrame::MAXSAMPLES * nchannels_per_block *
            4 * sizeof(SampleVector::Scalar);
        auto const threads_nullhypot = threads_1d;
        auto const blocks_nullhypot = blocks_1d;
        kernel_time_compute_nullhypot<<<blocks_nullhypot, threads_nullhypot, 
                                        sharedBytes, cudaStream.id()>>>(
            scratch.sample_values,
            scratch.sample_value_errors,
            scratch.useless_sample_values,
            scratch.chi2sNullHypot,
            scratch.sum0sNullHypot,
            scratch.sumAAsNullHypot,
            totalChannels
        );
        cudaCheck(hipGetLastError());

        unsigned int nchannels_per_block_makeratio = 10;
        unsigned int threads_makeratio = 45 * nchannels_per_block_makeratio;
        unsigned int blocks_makeratio = threads_makeratio > 45 * totalChannels
            ? 1
            : (totalChannels * 45 + threads_makeratio - 1) / threads_makeratio;
        int sharedBytesMakeRatio = 5 * threads_makeratio * sizeof(SampleVector::Scalar);
        kernel_time_compute_makeratio<<<blocks_makeratio, threads_makeratio,
                                        sharedBytesMakeRatio, cudaStream.id()>>>(
            scratch.sample_values,
            scratch.sample_value_errors,
            eventInputGPU.ids,
            scratch.useless_sample_values,
            scratch.pedestal_nums,
            configParameters.amplitudeFitParametersEB,
            configParameters.amplitudeFitParametersEE,
            configParameters.timeFitParametersEB,
            configParameters.timeFitParametersEE,
            scratch.sumAAsNullHypot,
            scratch.sum0sNullHypot,
            scratch.tMaxAlphaBetas,
            scratch.tMaxErrorAlphaBetas,
            scratch.accTimeMax,
            scratch.accTimeWgt,
            scratch.tcState,
            configParameters.timeFitParametersSizeEB, 
            configParameters.timeFitParametersSizeEE,
            configParameters.timeFitLimitsFirstEB,
            configParameters.timeFitLimitsFirstEE,
            configParameters.timeFitLimitsSecondEB,
            configParameters.timeFitLimitsSecondEE,
            totalChannels
        );
        cudaCheck(hipGetLastError());

        //
        //
        //
        auto const threads_findamplchi2 = threads_1d;
        auto const blocks_findamplchi2 = blocks_1d;
        int const sharedBytesFindAmplChi2 = 2 * threads_findamplchi2 * 
            sizeof(SampleVector::Scalar);
        kernel_time_compute_findamplchi2_and_finish<<<blocks_findamplchi2,
                                           threads_findamplchi2,
                                           sharedBytesFindAmplChi2, cudaStream.id()>>>(
            scratch.sample_values,
            scratch.sample_value_errors,
            eventInputGPU.ids,
            scratch.useless_sample_values,
            scratch.tMaxAlphaBetas,
            scratch.tMaxErrorAlphaBetas,
            scratch.accTimeMax,
            scratch.accTimeWgt,
            configParameters.amplitudeFitParametersEB,
            configParameters.amplitudeFitParametersEE,
            scratch.sumAAsNullHypot,
            scratch.sum0sNullHypot,
            scratch.chi2sNullHypot,
            scratch.tcState,
            scratch.ampMaxAlphaBeta,
            scratch.ampMaxError,
            scratch.timeMax,
            scratch.timeError,
            totalChannels
        );
        cudaCheck(hipGetLastError());
        
        //
        //
        //
        auto const threads_timecorr = 32;
        auto const blocks_timecorr = threads_timecorr > totalChannels
            ? 1 : (totalChannels + threads_timecorr-1) / threads_timecorr;
        kernel_time_correction_and_finalize<<<blocks_timecorr, threads_timecorr,
                                              0, cudaStream.id()>>>(
            eventOutputGPU.amplitude,
            eventInputGPU.digis,
            eventInputGPU.ids,
            conditions.timeBiasCorrections.EBTimeCorrAmplitudeBins,
            conditions.timeBiasCorrections.EETimeCorrAmplitudeBins,
            conditions.timeBiasCorrections.EBTimeCorrShiftBins,
            conditions.timeBiasCorrections.EETimeCorrShiftBins,
            scratch.timeMax,
            scratch.timeError,
            conditions.pedestals.rms_x12,
            conditions.timeCalibConstants.values,
            eventOutputGPU.jitter,
            eventOutputGPU.jitterError,
            eventOutputGPU.flags,
            conditions.timeBiasCorrections.EBTimeCorrAmplitudeBinsSize,
            conditions.timeBiasCorrections.EETimeCorrAmplitudeBinsSize,
            configParameters.timeConstantTermEB,
            configParameters.timeConstantTermEE,
            conditions.timeOffsetConstant.getEBValue(),
            conditions.timeOffsetConstant.getEEValue(),
            configParameters.timeNconstEB,
            configParameters.timeNconstEE,
            configParameters.amplitudeThreshEB,
            configParameters.amplitudeThreshEE,
            configParameters.outOfTimeThreshG12pEB,
            configParameters.outOfTimeThreshG12pEE,
            configParameters.outOfTimeThreshG12mEB,
            configParameters.outOfTimeThreshG12mEE,
            configParameters.outOfTimeThreshG61pEB,
            configParameters.outOfTimeThreshG61pEE,
            configParameters.outOfTimeThreshG61mEB,
            configParameters.outOfTimeThreshG61mEE,
            offsetForHashes,
            totalChannels
        );
        cudaCheck(hipGetLastError());
    }

        /*
    hipEventRecord(end_event, 0);
    hipEventSynchronize(end_event);
    float ms;
    hipEventElapsedTime(&ms, start_event, end_event);
    std::cout << "elapsed time = " << ms << std::endl;
    */
}

}}
