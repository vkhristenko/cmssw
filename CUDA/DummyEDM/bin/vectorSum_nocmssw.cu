#include "hip/hip_runtime.h"
#include <stdio.h>
#include <future>
#include <thread>
#include <chrono>
#include <iostream>
#include <iterator>
#include <cstring>

#define N 1000000
#define SIZE 100

__constant__ int factor = 1;

//
// NOTE: while loop is for the case when number of elements in the array exceeds the 
// number of blocks possible total in device to be launched
//
__global__ 
void vectorAdd(int *a, int *b, int *c) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    while (i < N) {
        c[i] = factor*(a[i] + b[i]);
        i += blockDim.x * gridDim.x;
    }
}

__global__
void matrixAdd(int **a,int **b, int**c) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    c[i][j] = a[i][j] + b[i][j];
}

#define PRINT(x) \
    std::cout << #x " = " << x << std::endl

void printImage(char* ptr) {
    for (auto i=0; i<SIZE; i++) {
        char cpyPtr[SIZE+1];
        std::memcpy((void*)cpyPtr, (void*)(ptr + SIZE*i), SIZE);
        cpyPtr[SIZE] = '\0';
        printf("%s\n", cpyPtr);
    }

    printf("\n");
}

int main(int argc, char** argv) {
    // start time
    auto startTime = std::chrono::high_resolution_clock::now();
    printf("Hello World\n");

    // get the number of devices
    int numDevices;
    hipGetDeviceCount(&numDevices);
    PRINT(numDevices);

    hipDeviceProp_t prop;
    for (auto i=0 ; i<numDevices; i++) {
        hipGetDeviceProperties(&prop, i);
        PRINT(prop.name);
        PRINT(prop.totalGlobalMem);
        PRINT(prop.sharedMemPerBlock);
        PRINT(prop.regsPerBlock);
        PRINT(prop.warpSize);
        PRINT(prop.memPitch);
        PRINT(prop.maxThreadsPerBlock);
        PRINT(prop.maxThreadsDim[0]);
        PRINT(prop.maxThreadsDim[1]);
        PRINT(prop.maxThreadsDim[2]);
        PRINT(prop.maxGridSize[0]);
        PRINT(prop.maxGridSize[1]);
        PRINT(prop.maxGridSize[2]);
        PRINT(prop.totalConstMem);
        PRINT(prop.major);
        PRINT(prop.minor);
        PRINT(prop.clockRate);
        PRINT(prop.textureAlignment);
        PRINT(prop.deviceOverlap);
        PRINT(prop.multiProcessorCount);
        PRINT(prop.kernelExecTimeoutEnabled);
        PRINT(prop.integrated);
        PRINT(prop.canMapHostMemory);
        PRINT(prop.computeMode);
        PRINT(prop.maxTexture1D);
        PRINT(prop.maxTexture2D[0]);
        PRINT(prop.maxTexture2D[1]);
        PRINT(prop.maxTexture3D[0]);
        PRINT(prop.maxTexture3D[1]);
        PRINT(prop.maxTexture3D[2]);
//        PRINT(prop.maxTexture2DArray[0]);
//        PRINT(prop.maxTexture2DArray[1]);
//        PRINT(prop.maxTexture2DArray[2]);
        PRINT(prop.concurrentKernels);
    }

    int h_a[N], h_b[N], h_c[N];
    int *d_a, *d_b, *d_c;

    hipMalloc(&d_a, N*sizeof(int));
    hipMalloc(&d_b, N*sizeof(int));
    hipMalloc(&d_c, N*sizeof(int));

    for (auto i=0; i<N; i++) {
        h_a[i ] = i;
        h_b[i] = i*i;
    }

    hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice);


    vectorAdd<<<(N+127)/128, 128>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost);

    for (auto i=0; i<N; i++) {
        if (i < 10)
            printf("c[%d] = %d\n", i, h_c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // stop time
    auto stopTime = std::chrono::high_resolution_clock::now();
    PRINT((stopTime - startTime).count());

    printf("Goodbye World\n");
}
