#include "hip/hip_runtime.h"
#include "TestGPU/Dummy/interface/gpu_kernels.h"

#include <stdio.h>

namespace testgpu {

//
// Vector Addition Kernel
//
template<typename T>
__global__
void vectorAdd(T *a, T *b, T *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (size_t j=0; j<1000000; j++)
        c[i] = a[i] + b[i];
}

//
// Vector Addition Kernel Wrapper
//
template<typename T>
void wrapperVectorAdd(T* d_a, T* d_b, T* d_c, hipStream_t stream, int N) {
    int threadsPerBlock {256};
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_a, d_b, d_c);
}

//
// Macros to simplify the template instantiation
//
#define WRAPPERVECTORADD(TYPE) \
    template void wrapperVectorAdd<TYPE>(TYPE*, TYPE*, TYPE*, hipStream_t, int)

//
// NOTE:
// -----
// We have to instantiate tempaltes explicitly given that kernels will be compiled 
// as separate compilation units and linked afterwards.
// 
//
WRAPPERVECTORADD(int);
WRAPPERVECTORADD(float);
WRAPPERVECTORADD(double);
WRAPPERVECTORADD(long);

//
// Standalone function that allocates/copies/launches/frees and prints the results
//
void launch_on_gpu() {
    int const NUM_VALUES = 10000;
    printf("start launch_on_gpu\n");
    int h_a[NUM_VALUES], h_b[NUM_VALUES], h_c[NUM_VALUES];
    for (auto i=0; i<NUM_VALUES; i++) {
        h_a[i] = i;
        h_b[i] = i*i;
    }

    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, NUM_VALUES*sizeof(int));
    hipMalloc(&d_b, NUM_VALUES*sizeof(int));
    hipMalloc(&d_c, NUM_VALUES*sizeof(int));

    hipMemcpy(d_a, h_a, NUM_VALUES*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, NUM_VALUES*sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock {256};
    int blocksPerGrid = (NUM_VALUES + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, NUM_VALUES*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for (auto i=0; i<10; i++) {
        printf("c[%d] = %d\n", i, h_c[i]);
    }

    printf("\n");
    printf("stop launch_on_gpu\n");
}

}
